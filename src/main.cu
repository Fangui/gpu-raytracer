#include "hip/hip_runtime.h"
#include <fstream>
#include <vector>
#include <omp.h>
#include <chrono>

#include "parse.hh"
#include "vector.hh"
#include "device.hh"

using namespace std::chrono;

__global__ void render(Pixel *d_vect, KdNodeGpu *d_tree, Material *d_materials,
                       Vector *a_light, Light *d_lights, size_t d_lights_len, Vector *d_u, Vector *d_v,
                       Vector *d_center, Vector *d_cam_pos, 
                       unsigned width, unsigned height)
{
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = blockDim.y * blockIdx.y + threadIdx.y;

    if (i >= width || j >= height)
        return;

    Vector o = *d_u * (j - static_cast<int>(height) / 2);
    Vector b = *d_v * (i - static_cast<int>(width) / 2);

    o += *d_center;
    o += b;

    Vector dir = (o - *d_cam_pos).norm_inplace();
    Ray ray(*d_cam_pos, dir);

    d_vect[i * width + j] = direct_light(d_tree, ray, d_materials, a_light, d_lights, d_lights_len);
}

int main(int argc, char *argv[])
{
    std::string path_scene;
    std::string out_file = "out";

    if (argc > 1)
        path_scene = argv[1];
    else
    {
        std::cerr << "Usage: ./main <scene> <outfile>\n";
        return 1;
    }

    if (argc > 2)
        out_file = argv[2];

    Scene scene = parse_scene(path_scene);

    Vector u_n = scene.cam_u.norm_inplace();
    Vector v_n = scene.cam_v.norm_inplace();
    Vector w = v_n.cross_product(u_n);

    float val = tan(scene.fov * M_PI / 360);
    float l = scene.width / 2;
    l /= val;
    Vector center = scene.cam_pos + (w * l); // center

    std::vector<Triangle> vertices;
    for (const auto& name : scene.objs)
      obj_to_vertices(name, scene.mat_names, vertices, scene);

    auto t1 = high_resolution_clock::now();
    auto tree = KdTree(vertices.begin(), vertices.end());
    std::cout << tree.size() << std::endl;

    KdNodeGpu *d_tree = upload_kd_tree(tree, vertices);
    auto t2 = high_resolution_clock::now();
    std::cout << "Time to build tree: " << duration_cast<duration<double>>(t2 - t1).count()  << "s\n";

    Pixel *d_vect;
    Vector *d_u;
    Vector *d_v;
    Vector *d_center;
    Vector *d_cam_pos;
    Material *d_materials;
    Vector *a_light;
    Light *d_lights;

   cudaCheckError(hipMalloc(&d_materials, scene.mat_names.size() * sizeof(Material)));
   cudaCheckError(hipMemcpy(d_materials, scene.materials, 
               scene.mat_names.size() * sizeof(Material), hipMemcpyHostToDevice));

   cudaCheckError(hipMalloc(&d_vect, scene.width * scene.height * sizeof(*d_vect)));
   cudaCheckError(hipMalloc(&d_u, sizeof(struct Vector)));
   cudaCheckError(hipMalloc(&d_v, sizeof(struct Vector)));
   cudaCheckError(hipMalloc(&d_center, sizeof(struct Vector)));
   cudaCheckError(hipMalloc(&d_cam_pos, sizeof(struct Vector)));
   Pixel *vect = new Pixel[scene.width * scene.height];

   cudaCheckError(hipMalloc(&a_light, sizeof(Vector)));
   cudaCheckError(hipMemcpy(a_light, &scene.a_light, sizeof(Vector), hipMemcpyHostToDevice));

   cudaCheckError(hipMalloc(&d_lights, sizeof(Light) * scene.lights.size()));
   cudaCheckError(hipMemcpy(d_lights, scene.lights.data(), 
                  sizeof(Light) * scene.lights.size(), hipMemcpyHostToDevice));

   cudaCheckError(hipMemcpy(d_u, &u_n, sizeof(struct Vector), hipMemcpyHostToDevice));
   cudaCheckError(hipMemcpy(d_v, &v_n, sizeof(struct Vector), hipMemcpyHostToDevice));
   cudaCheckError(hipMemcpy(d_center, &center, sizeof(struct Vector), hipMemcpyHostToDevice));
   cudaCheckError(hipMemcpy(d_cam_pos, &scene.cam_pos, 
                             sizeof(struct Vector), hipMemcpyHostToDevice));

    constexpr int tx = 15; 
    constexpr int ty = 32;

    dim3 dim_block(scene.width / tx + (scene.width % tx != 0),
                   scene.height / ty + (scene.height % ty != 0));
    dim3 dim_thread(tx, ty);

    t1 = high_resolution_clock::now();
    render<<<dim_block, dim_thread >>>(d_vect, d_tree, d_materials, a_light, d_lights, scene.lights.size(), d_u, d_v, d_center, d_cam_pos,
                                      scene.width, scene.height);


    t2 = high_resolution_clock::now();
    std::cout << "Time to ray tracer: " << duration_cast<duration<double>>(t2 - t1).count() << "s\n";
    cudaCheckError(hipMemcpy(vect, d_vect, scene.width * scene.height * sizeof(*d_vect),
                   hipMemcpyDeviceToHost));
    auto t3 = high_resolution_clock::now();

    std::cout << "Time to memcpy: " << duration_cast<duration<double>>(t3 - t2).count() << "s\n";

    hipFree(d_materials);
    hipFree(a_light);
    hipFree(d_lights);
    hipFree(d_vect);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_center);
    hipFree(d_cam_pos);

    write_ppm(out_file + ".ppm", vect, scene.width, scene.height);
    delete[] vect;
}
